#include "hip/hip_runtime.h"
#include <cmath>
#include <memory>
#include <cstdint>
#include <iostream>
#include <hip/hip_runtime.h>

#include "Image.hpp"

// Small helper functions
template <typename _T>
inline _T Clamp(const _T& x, const _T& min, const _T& max) noexcept {
  if (x > max) return max;
  if (x < min) return min;

  return x;
}

__global__ void RayTrace(Colorf32* pFloatImage, const size_t width, const size_t height) {
  // Calculate the thread's (X, Y) location
  const size_t pixelX = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t pixelY = threadIdx.y + blockIdx.y * blockDim.y;

  // Bounds check
  if (pixelX >= width || pixelY >= height) return;

  // Determine the pixel's index into the image buffer
  const size_t index = pixelX + pixelY * width;
  

  
  *(pFloatImage + index) = Colorf32{pixelX / (float)width, pixelY/(float)height, 0.5f, 1.f};
}

void RayTraceScene(Image& cpuOutputImage) {
  const size_t f32BufferSize = cpuOutputImage.GetPixelCount() * sizeof(Colorf32);

  // allocate raw GPU Buffer
  float* gpuImageBuffer;
  hipMallocManaged(&gpuImageBuffer, f32BufferSize);

  // calculate the __
  const size_t nThreadsPerBlock = 512; // 32*32*32 (32*32 warps of 32 threads)
  const size_t nBlocks          = std::ceil(cpuOutputImage.GetPixelCount() / static_cast<float>(nThreadsPerBlock));

  const dim3 dimBlock = dim3(16, 16);
  const dim3 dimGrid  = dim3(std::ceil(cpuOutputImage.GetWidth()  / static_cast<float>(dimBlock.x)),
                             std::ceil(cpuOutputImage.GetHeight() / static_cast<float>(dimBlock.y)));

  // trace rays through each pixel
  RayTrace<<<dimGrid, dimBlock>>>((Colorf32*)gpuImageBuffer, cpuOutputImage.GetWidth(), cpuOutputImage.GetHeight());
  
  // wait for the job to finish
  hipDeviceSynchronize();

  // copy the gpu buffer to a new cpu buffer
  std::unique_ptr<float[]> cpuF32Buffer = std::make_unique<float[]>(cpuOutputImage.GetPixelCount() * 4);
  hipMemcpy(cpuF32Buffer.get(), gpuImageBuffer, f32BufferSize, hipMemcpyDeviceToHost);

  // free the GPU image buffer
  hipFree(gpuImageBuffer);

  // write the image data to the image
  for (std::uint32_t i = 0; i < cpuOutputImage.GetPixelCount(); i++) {
    Coloru8& pixel = cpuOutputImage(i);
    pixel.r = static_cast<std::uint8_t>(Clamp(cpuF32Buffer[i * 4 + 0] * 255.f, 0.f, 255.f));
    pixel.g = static_cast<std::uint8_t>(Clamp(cpuF32Buffer[i * 4 + 1] * 255.f, 0.f, 255.f));
    pixel.b = static_cast<std::uint8_t>(Clamp(cpuF32Buffer[i * 4 + 2] * 255.f, 0.f, 255.f));
    pixel.a = static_cast<std::uint8_t>(Clamp(cpuF32Buffer[i * 4 + 3] * 255.f, 0.f, 255.f));
  }
}