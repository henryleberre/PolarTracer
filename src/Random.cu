#ifndef __POLAR_TRACER__RANDOM_CU
#define __POLAR_TRACER__RANDOM_CU

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "Enums.cu"
#include "Vector.cu"

__device__ float RandomFloat(hiprandState_t* const randState) noexcept {
    return hiprand_uniform(randState);
}

__device__ inline Vec4f32 Random3DUnitVector(hiprandState_t* const randState) noexcept {
    return Vec4f32::Normalized3D(Vec4f32(2.0f * RandomFloat(randState) - 1.0f,
        2.0f * RandomFloat(randState) - 1.0f,
        2.0f * RandomFloat(randState) - 1.0f,
        0.f));
}

#endif // __POLAR_TRACER__RANDOM_CU