#include "hip/hip_runtime.h"
#include <chrono>
#include <limits>
#include <ostream>
#include <cassert>
#include <stdio.h>
#include <optional>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

//TODO:
// + Check that the intersection normals always point in the correct direction
// + Fresnel
// + Obj Mesh loading

#define EPSILON (float)0.0001f
#define MAX_REC (10)
#define SPP     (10)

__device__ float RandomFloat(hiprandState_t* const randState) noexcept {
    return hiprand_uniform(randState);
}

template <typename T>
__host__ __device__ inline T Clamp(const T& x, const T& min, const T& max) noexcept {
    return (x > max) ? max : ((x < min) ? min : x);
}

template <typename T>
__host__ __device__ inline T Swap(T& a, T& b) noexcept {
    const T tmp = a;
    a = b;
    b = tmp;
}

// The "Device" enum class represents the devices from which
// memory can be accessed. This is necessary because the cpu can't
// read/write directly from/to the GPU's memory and conversely.
enum class Device { CPU, GPU }; // Device

// The Pointer<typename T, Device D> class represents a C++ Pointer of base type
// T that is accessible from the device D (view enum Device).
template <typename T, Device D>
class Pointer {
private:
    T* m_raw = nullptr;

public:
    Pointer() = default;

    __host__ __device__ inline Pointer(T* const p)              noexcept { this->SetPointer(p); }
    __host__ __device__ inline Pointer(const Pointer<T, D>& o) noexcept { this->SetPointer(o); }

    __host__ __device__ inline void SetPointer(T* const p)              noexcept { this->m_raw = p; }
    __host__ __device__ inline void SetPointer(const Pointer<T, D>& o) noexcept { this->SetPointer(o.m_raw); }

    __host__ __device__ inline T*& GetPointer()       noexcept { return this->m_raw; }
    __host__ __device__ inline T*  GetPointer() const noexcept { return this->m_raw; }

    template <typename U>
    __host__ __device__ inline Pointer<U, D> AsPointerTo() const noexcept {
        return Pointer<U, D>(reinterpret_cast<U*>(this->m_raw));
    }

    __host__ __device__ inline void operator=(T* const p)                      noexcept { this->SetPointer(p); }
    __host__ __device__ inline void operator=(const Pointer<T, D>& o) noexcept { this->SetPointer(o); }

    __host__ __device__ inline operator T*& ()       noexcept { return this->m_raw; }
    __host__ __device__ inline operator T*  () const noexcept { return this->m_raw; }

    __host__ __device__ inline std::conditional_t<std::is_same_v<T, void>, int, T>& operator[](const size_t i) noexcept {
        static_assert(!std::is_same_v<T, void>, "Can't Index A Pointer To A Void");
        return *(this->m_ptr + i);
    }

    __host__ __device__ inline const std::conditional_t<std::is_same_v<T, void>, int, T>& operator[](const size_t i) const noexcept {
        static_assert(!std::is_same_v<T, void>, "Can't Index A Pointer To A Void");
        return *(this->m_ptr + i);
    }

    __host__ __device__ inline T* operator->() const noexcept { return this->m_raw; }
}; // Pointer<T>

// Some aliases for the Pointer<T, D> class.
template <typename T>
using CPU_Ptr = Pointer<T, Device::CPU>;

template <typename T>
using GPU_Ptr = Pointer<T, Device::GPU>;

// Memory Allocation
template <typename T, Device D>
inline Pointer<T, D> AllocateSize(const size_t size) noexcept {
    if constexpr (D == Device::CPU) {
        return CPU_Ptr<T>(reinterpret_cast<T*>(std::malloc(size)));
    } else {
        T* p;
        hipMalloc(reinterpret_cast<void**>(&p), size);
        return GPU_Ptr<T>(p);
    }
}

template <typename T, Device D>
inline Pointer<T, D> AllocateCount(const size_t count) noexcept {
    return AllocateSize<T, D>(count * sizeof(T));
}

template <typename T, Device D>
inline Pointer<T, D> AllocateSingle() noexcept {
    return AllocateSize<T, D>(sizeof(T));
}

// Memory Deallocation

template <typename T, Device D>
inline void Free(const Pointer<T, D>& p) noexcept {
    if constexpr (D == Device::CPU) {
        std::free(p);
    } else {
        hipFree(p.template AsPointerTo<void>());
    }
}

// Copying Memory
// These functions take as arguments objects of type _PTR<T, D>
// such as Pointer<T, D>, ArrayView<T, D> or Array<T, D>.
// The object _PTR<T, D> has to be convertible to a raw pointer of
// base type T accessible by the device D.

template <template<typename, Device> typename _PTR_DST, typename T_DST, Device D_DST,
          template<typename, Device> typename _PTR_SRC, typename T_SRC, Device D_SRC>
inline void CopySize(const _PTR_DST<T_DST, D_DST>& dst,
                     const _PTR_SRC<T_SRC, D_SRC>& src,
                     const size_t size) noexcept
{
    static_assert(std::is_same_v<T_DST, T_SRC>, "Incompatible Source And Destination Raw Pointer Types");

    hipMemcpyKind memcpyKind;

    if constexpr (D_SRC == Device::CPU && D_DST == Device::CPU) {
        memcpyKind = hipMemcpyKind::hipMemcpyHostToHost;
        hipMemcpy(dst, src, size, memcpyKind);
    } else if constexpr (D_SRC == Device::GPU && D_DST == Device::GPU) {
        memcpyKind = hipMemcpyKind::hipMemcpyDeviceToDevice;
        hipMemcpy(dst, src, size, memcpyKind);
    } else if constexpr (D_SRC == Device::CPU && D_DST == Device::GPU) {
        memcpyKind = hipMemcpyKind::hipMemcpyHostToDevice;
        hipMemcpy(dst, src, size, memcpyKind);
    } else if constexpr (D_SRC == Device::GPU && D_DST == Device::CPU) {
        memcpyKind = hipMemcpyKind::hipMemcpyDeviceToHost;
        hipMemcpy(dst, src, size, memcpyKind);
    } else { static_assert(1 == 1, "Incompatible Destination and Source Arguments"); }
}

template <template<typename, Device> typename _PTR_DST, typename T_DST, Device D_DST,
         template<typename, Device> typename _PTR_SRC, typename T_SRC, Device D_SRC>
inline void CopyCount(const _PTR_DST<T_DST, D_DST>& dst,
                                          const _PTR_SRC<T_SRC, D_SRC>& src,
                                          const size_t count) noexcept
{
    static_assert(std::is_same_v<T_DST, T_SRC>, "Incompatible Source And Destination Raw Pointer Types");

    CopySize(dst, src, count * sizeof(T_DST));
}

template <template<typename, Device> typename _PTR_DST, typename T_DST, Device D_DST,
          template<typename, Device> typename _PTR_SRC, typename T_SRC, Device D_SRC>
inline void CopySingle(const _PTR_DST<T_DST, D_DST>& dst,
                                           const _PTR_SRC<T_SRC, D_SRC>& src) noexcept
{
    static_assert(std::is_same_v<T_DST, T_SRC>, "Incompatible Source And Destination Raw Pointer Types");

    CopySize(dst, src, sizeof(T_DST));
}


// The UniquePointer<typename T, Device D> class represents a C++ Pointer
// of base type T whose memory is owned and managed by this class. As a result,
// when this class is destroyed or it's owning memory location changes, it will
// free the memory it owned.
template <typename T, Device D>
class UniquePointer {
private:
    Pointer<T, D> m_ptr;

public:
    inline UniquePointer() noexcept {
        this->Free();
        this->m_ptr = nullptr;
    }

    template <typename... _ARGS>
    inline UniquePointer(const _ARGS&... args) noexcept {
        this->Free();
        this->m_ptr = new (AllocateSingle<T, D>()) T(std::forward<_ARGS>(args)...);
    }

    inline UniquePointer(const Pointer<T, D>& o)  noexcept {
        this->Free();
        this->m_ptr = o;
    }

    inline UniquePointer(UniquePointer<T, D>&& o) noexcept {
        this->Free();
        this->m_ptr = o.m_ptr;
        o.m_ptr = nullptr;
    }

    inline void Free() const noexcept {
        // Since we use placement new, we have to call T's destructor ourselves
        this->m_ptr->~T();

        // and then free the memory
        ::Free(this->m_ptr);
    }

    inline ~UniquePointer() noexcept { this->Free(); }

    inline UniquePointer<T, D>& operator=(const Pointer<T, D>& o)  noexcept {
        this->Free();
        this->m_ptr = o;
        return *this;
    }

    inline UniquePointer<T, D>& operator=(UniquePointer<T, D>&& o) noexcept {
        this->Free();
        this->m_ptr = o;
        o.m_ptr = nullptr;
        return *this;
    }

    __host__ __device__ inline const Pointer<T, D>& GetPointer() const noexcept { return this->m_ptr; }

    __host__ __device__ inline operator const Pointer<T, D>& () const noexcept { return this->m_ptr; }

    __host__ __device__ inline operator T* () const noexcept { return this->m_ptr; }

    __host__ __device__ inline T* operator->() const noexcept { return this->m_ptr; }

    __host__ __device__ inline       T& operator[](const size_t i)       noexcept { return *(this->m_ptr + i); }
    __host__ __device__ inline const T& operator[](const size_t i) const noexcept { return *(this->m_ptr + i); }


    UniquePointer(const UniquePointer<T, D>& o) = delete;
    UniquePointer<T, D>& operator=(const UniquePointer<T, D>& o) = delete;
}; // UniquePointer<T, D>

// Some aliases for the UniquePointer<T, D> class.
template <typename T>
using CPU_UniquePtr = UniquePointer<T, Device::CPU>;

template <typename T>
using GPU_UniquePtr = UniquePointer<T, Device::GPU>;

// The ArrayView<typename T, Device D> class represents a
// contiguous allocation of memory on the device D of elements of type
// T. It is defined by a starting memory address and a count of elements
// of type T following the address.
template <typename T, Device D>
class ArrayView {
private:
    size_t m_count = 0;
    Pointer<T, D> m_pBegin;

protected:
    __host__ __device__ inline void SetPointer(const Pointer<T, D>& pBegin) noexcept { this->m_pBegin = pBegin; }
    __host__ __device__ inline void SetCount(const size_t count)                    noexcept { this->m_count = count; }

    __host__ __device__ inline operator T*& () noexcept { return this->m_pBegin; }

public:
    ArrayView() = default;

    inline ArrayView(const Pointer<T, D>& pBegin, const size_t count) noexcept
        : m_pBegin(pBegin), m_count(count)
    {  }

    __host__ __device__ inline const Pointer<T, D>& GetPointer() const noexcept { return this->m_pBegin; }
    __host__ __device__ inline const size_t&        GetCount()   const noexcept { return this->m_count; }

    __host__ __device__ inline operator const Pointer<T, D>& () const noexcept { return this->m_pBegin; }

    __host__ __device__ inline operator T* () const noexcept { return this->m_pBegin; }

    __host__ __device__ inline       T& operator[](const size_t i)       noexcept { return *(this->m_pBegin + i); }
    __host__ __device__ inline const T& operator[](const size_t i) const noexcept { return *(this->m_pBegin + i); }

    ~ArrayView() = default;
}; // ArrayView<T, D>

// Some aliases for the ArrayView<T, D> class.
template <typename T>
using CPU_ArrayView = ArrayView<T, Device::CPU>;

template <typename T>
using GPU_ArrayView = ArrayView<T, Device::GPU>;


// The Array<typename T, Device D> is essentialy a
// ArrayView<T, D> who owns the memory it represents.
template <typename T, Device D>
class Array : public ArrayView<T, D> {
public:
    inline Array() noexcept = default;

    inline Array(const size_t count) noexcept {
        this->SetCount(count);
        this->SetPointer(AllocateCount<T, D>(count));
    }

    template <Device D_O>
    inline Array(const Array<T, D_O>& o) noexcept
        : Array(o.GetCount())
    {
        CopyCount(*this, o, this->GetCount());
    }

    inline Array(Array<T, D>&& o) noexcept
    {
        this->SetCount(o.GetCount());
        o.SetCount(0);
        this->SetPointer(o.GetPointer());
        o.SetPointer((T*)nullptr);
    }

    inline Array<T, D>& operator=(Array<T, D>&& o) noexcept {
        this->SetCount(o.GetCount());
        o.SetCount(0);
        this->SetPointer(o.GetPointer());
        o.SetPointer((T*)nullptr);

        return *this;
    }

    inline void Reserve(const size_t count) noexcept {
        const auto newCount = this->GetCount() + count;
        const auto newBegin = AllocateCount<T, D>(newCount);

        CopyCount(newBegin, this->GetPointer(), this->GetCount());
        Free(this->GetPointer());

        this->SetPointer(newBegin);
        this->SetCount(newCount);
    }

    inline ~Array() noexcept {
        Free(this->GetPointer());
    }
}; // Array<T, D>

// Some aliases for the Array<T, D> class.
template <typename T>
using CPU_Array = Array<T, Device::CPU>;

template <typename T>
using GPU_Array = Array<T, Device::GPU>;


template <typename T>
struct Vec4 {
    T x, y, z, w;

    template <typename _H = T, typename _V = T, typename _K = T, typename _Q = T>
    __host__ __device__ inline Vec4(const _H& x = 0, const _V& y = 0, const _K& z = 0, const _Q& w = 0) noexcept {
        this->x = static_cast<T>(x); this->y = static_cast<T>(y); this->z = static_cast<T>(z); this->w = static_cast<T>(w);
    }

    __host__ __device__ inline void Clamp(const float min, const float max) noexcept {
        this->x = ::Clamp(this->x, min, max);
        this->y = ::Clamp(this->y, min, max);
        this->z = ::Clamp(this->z, min, max);
        this->w = ::Clamp(this->w, min, max);
    }

    __host__ __device__ inline float GetLength3D() const noexcept { return sqrt(this->x * this->x + this->y * this->y + this->z * this->z); }
    __host__ __device__ inline float GetLength4D() const noexcept { return sqrt(this->x * this->x + this->y * this->y + this->z * this->z + this->w * this->w); }

    __host__ __device__ inline void Normalize3D() noexcept { this->operator/=(this->GetLength3D()); }
    __host__ __device__ inline void Normalize4D() noexcept { this->operator/=(this->GetLength4D()); }

    template <typename _U>
    __host__ __device__ inline Vec4<T>& operator+=(const Vec4<_U>& o) noexcept {
        this->x += o.x; this->y += o.y; this->z += o.z; this->w += o.w;
        return *this;
    }

    template <typename _U>
    __host__ __device__ inline Vec4<T>& operator-=(const Vec4<_U>& o) noexcept {
        this->x -= o.x; this->y -= o.y; this->z -= o.z; this->w -= o.w;
        return *this;
    }

    template <typename _U>
    __host__ __device__ inline Vec4<T>& operator*=(const Vec4<_U>& o) noexcept {
        this->x *= o.x; this->y *= o.y; this->z *= o.z; this->w *= o.w;
        return *this;
    }

    template <typename _U>
    __host__ __device__ inline Vec4<T>& operator/=(const Vec4<_U>& o) noexcept {
        this->x /= o.x; this->y /= o.y; this->z /= o.z; this->w /= o.w;
        return *this;
    }


    template <typename _U>
    __host__ __device__ inline Vec4<T>& operator+=(const _U& n) noexcept {
        this->x += n; this->y += n; this->z += n; this->w += n;
        return *this;
    }

    template <typename _U>
    __host__ __device__ inline Vec4<T>& operator-=(const _U& n) noexcept {
        this->x -= n; this->y -= n; this->z -= n; this->w -= n;
        return *this;
    }

    template <typename _U>
    __host__ __device__ inline Vec4<T>& operator*=(const _U& n) noexcept {
        this->x *= n; this->y *= n; this->z *= n; this->w *= n;
        return *this;
    }

    template <typename _U>
    __host__ __device__ inline Vec4<T>& operator/=(const _U& n) noexcept {
        this->x /= n; this->y /= n; this->z /= n; this->w /= n;
        return *this;
    }

    static __host__ __device__ inline float DotProduct3D(const Vec4<T>& a, const Vec4<T>& b) noexcept {
        return a.x * b.x + a.y * b.y + a.z * b.z;
    }

    static __host__ __device__ inline float DotProduct4D(const Vec4<T>& a, const Vec4<T>& b) noexcept {
        return a.x * b.x + a.y * b.y + a.z * b.z + a.w * b.w;
    }

    static __host__ __device__ inline Vec4<T> Normalized3D(Vec4<T> v) noexcept {
        v.Normalize3D();
        return v;
    }

    static __host__ __device__ inline Vec4<T> Normalized4D(Vec4<T> v) noexcept {
        v.Normalize4D();
        return v;
    }

    static __host__ __device__ inline Vec4<T> Reflected3D(const Vec4<T>& inDirection, const Vec4<T>& normal) noexcept {
        return inDirection - 2 * Vec4<T>::DotProduct3D(inDirection, normal) * normal;
    }

    static __host__ __device__ inline Vec4<T> CrossProduct3D(const Vec4<T>& a, const Vec4<T>& b) noexcept {
        return Vec4<T>{
            a.y*b.z-a.z*b.y,
            a.z*b.x-a.x*b.z,
            a.x*b.y-a.y*b.x,
            0
        };
    }

    static __host__ __device__ inline Vec4<T> Clamped(Vec4<T> v, const float min, const float max) noexcept {
        v.Clamp(min, max);
        return v;
    }
}; // Vec4<T>

template <typename T>
__host__ __device__ inline Vec4<T> Refract(const Vec4<T>& in, Vec4<T> n, const float ior) noexcept {
    float cosi = Clamp(Vec4<T>::DotProduct3D(in, n), -1.f, 1.f); 
    float etai = 1, etat = ior; 
    if (cosi < 0) {
        cosi = -cosi;
    } else {
        Swap(etai, etat);
        n *= -1;
    }

    float eta = etai / etat; 
    float k = 1 - eta * eta * (1 - cosi * cosi); 

    return (k < 0) ? Vec4f32(0, 0, 0, 0) : (eta * in + (eta * cosi - sqrtf(k)) * n); 
}

template <typename T, typename _U>
__host__ __device__ inline auto operator+(const Vec4<T>& a, const Vec4<_U>& b) noexcept -> Vec4<decltype(a.x + b.x)> {
    return Vec4<decltype(a.x + b.x)>{ a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w };
}

template <typename T, typename _U>
__host__ __device__ inline auto operator-(const Vec4<T>& a, const Vec4<_U>& b) noexcept -> Vec4<decltype(a.x - b.x)> {
    return Vec4<decltype(a.x - b.x)>{ a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w };
}

template <typename T, typename _U>
__host__ __device__ inline auto operator*(const Vec4<T>& a, const Vec4<_U>& b) noexcept -> Vec4<decltype(a.x* b.x)> {
    return Vec4<decltype(a.x* b.x)>{ a.x* b.x, a.y* b.y, a.z* b.z, a.w* b.w };
}

template <typename T, typename _U>
__host__ __device__ inline auto operator/(const Vec4<T>& a, const Vec4<_U>& b) noexcept -> Vec4<decltype(a.x / b.x)> {
    return Vec4<decltype(a.x / b.x)>{ a.x / b.x, a.y / b.y, a.z / b.z, a.w / b.w };
}


template <typename T, typename _U>
__host__ __device__ inline auto operator+(const Vec4<T>& a, const _U& n) noexcept -> Vec4<decltype(a.x + n)> {
    return Vec4<decltype(a.x + n)>{ a.x + n, a.y + n, a.z + n, a.w + n };
}

template <typename T, typename _U>
__host__ __device__ inline auto operator-(const Vec4<T>& a, const _U& n) noexcept -> Vec4<decltype(a.x - n)> {
    return Vec4<decltype(a.x - n)>{ a.x - n, a.y - n, a.z - n, a.w - n };
}

template <typename T, typename _U>
__host__ __device__ inline auto operator*(const Vec4<T>& a, const _U& n) noexcept -> Vec4<decltype(a.x* n)> {
    return Vec4<decltype(a.x* n)>{ a.x* n, a.y* n, a.z* n, a.w* n };
}

template <typename T, typename _U>
__host__ __device__ inline auto operator/(const Vec4<T>& a, const _U& n) noexcept -> Vec4<decltype(a.x / n)> {
    return Vec4<decltype(a.x / n)>{ a.x / n, a.y / n, a.z / n, a.w / n };
}


template <typename T, typename _U>
__host__ __device__ inline auto operator+(const _U& n, const Vec4<T>& a) { return a * n; }

template <typename T, typename _U>
__host__ __device__ inline auto operator-(const _U& n, const Vec4<T>& a) { return a * n; }

template <typename T, typename _U>
__host__ __device__ inline auto operator*(const _U& n, const Vec4<T>& a) { return a * n; }

template <typename T, typename _U>
__host__ __device__ inline auto operator/(const _U& n, const Vec4<T>& a) { return a * n; }

template <typename T>
std::ostream& operator<<(std::ostream& stream, const Vec4<T>& v) noexcept {
    stream << '(' << v.x << ", " << v.y << ", " << v.z << ", " << v.w << ')';

    return stream;
}

typedef Vec4<std::uint8_t> Coloru8;
typedef Vec4<float>        Colorf32;
typedef Vec4<float>        Vec4f32;


__device__ inline Vec4f32 Random3DUnitVector(hiprandState_t* const randState) noexcept {
    return Vec4f32::Normalized3D(Vec4f32(2.0f * RandomFloat(randState) - 1.0f,
        2.0f * RandomFloat(randState) - 1.0f,
        2.0f * RandomFloat(randState) - 1.0f,
        0.f));
}

template <typename T, Device D>
class Image {
private:
    std::uint16_t m_width = 0;
    std::uint16_t m_height = 0;
    std::uint32_t m_nPixels = 0;

    Array<T, D> m_pArray;

public:
    Image() = default;

    inline Image(const std::uint16_t width, const std::uint16_t height) noexcept
        : m_width(width),
        m_height(height),
        m_nPixels(static_cast<std::uint32_t>(width)* height),
        m_pArray(Array<T, D>(this->m_nPixels))
    { }

    __host__ __device__ inline std::uint16_t GetWidth()      const noexcept { return this->m_width; }
    __host__ __device__ inline std::uint16_t GetHeight()     const noexcept { return this->m_height; }
    __host__ __device__ inline std::uint32_t GetPixelCount() const noexcept { return this->m_nPixels; }

    __host__ __device__ inline Pointer<T, D> GetPtr() const noexcept { return this->m_pArray; }

    __host__ __device__ inline       T& operator()(const size_t i)       noexcept { return this->m_pArray[i]; }
    __host__ __device__ inline const T& operator()(const size_t i) const noexcept { return this->m_pArray[i]; }

    __host__ __device__ inline       T& operator()(const size_t x, const size_t y)       noexcept { return this->m_pArray[y * this->m_width + this->m_height]; }
    __host__ __device__ inline const T& operator()(const size_t x, const size_t y) const noexcept { return this->m_pArray[y * this->m_width + this->m_height]; }

}; // Image

void SaveImage(const Image<Coloru8, Device::CPU>& image, const std::string& filename) noexcept {
    const std::string fullFilename = filename + ".pam";

    // Open
    FILE* fp = std::fopen(fullFilename.c_str(), "wb");

    if (fp) {
        // Header
        std::fprintf(fp, "P7\nWIDTH %d\nHEIGHT %d\nDEPTH 4\nMAXVAL 255\nTUPLTYPE RGB_ALPHA\nENDHDR\n", image.GetWidth(), image.GetHeight());

        // Write Contents
        std::fwrite(image.GetPtr(), image.GetPixelCount() * sizeof(Coloru8), 1u, fp);

        // Close
        std::fclose(fp);
    }
}

struct Camera {
    float   fov;
    Vec4f32 position;
}; // Camera

struct RenderParams {
    size_t width = 0;
    size_t height = 0;

    Camera camera;
};

struct Intersection;

struct Ray {
    Vec4f32 origin;
    Vec4f32 direction;

    template <typename _T_OBJ>
    __device__ Intersection Intersects(const _T_OBJ& obj) const noexcept;
}; // Ray

struct Material {
    Colorf32 diffuse   = { 0.f, 0.f, 0.f, 1.f };
    Colorf32 emittance = { 0.f, 0.f, 0.f, 1.f };

    float reflectance  = 0.f; // the sum of these values should be less than or equal to 1
    float transparency = 0.f; // the sum of these values should be less than or equal to 1

    float roughness = 0.f;

    float index_of_refraction = 1.f;
}; // Material

struct ObjectBase {
    Material material;
};

struct Sphere : ObjectBase {
    Vec4f32 center;
    float   radius;
}; // Sphere

struct Plane : ObjectBase {
    Vec4f32 position; // Any Point On The Plane
    Vec4f32 normal;   // Normal To The Surface
}; // Plane

struct Triangle : ObjectBase {
    Vec4f32 p0; // Position of the 1st vertex
    Vec4f32 p1; // Position of the 2nd vertex
    Vec4f32 p2; // Position of the 3rd vertex
};

struct Intersection {
    Ray      inRay;     // incoming ray
    float            t; // distance from the ray's origin to intersection point
    Vec4f32  location;  // intersection location
    Vec4f32  normal;    // normal at intersection point
    Material material;  // the material that the intersected object is made of

    __device__ __host__ static inline Intersection MakeNullIntersection(const Ray& ray) noexcept {
        return Intersection{ray, FLT_MAX};
    }
}; // Intersection

template <>
__device__ Intersection Ray::Intersects(const Sphere& sphere) const noexcept {
    const float radius2 = sphere.radius * sphere.radius;

    const Vec4f32 L = sphere.center - this->origin;
    const float   tca = Vec4f32::DotProduct3D(L, this->direction);
    const float   d2 = Vec4f32::DotProduct3D(L, L) - tca * tca;

    if (d2 > radius2)
        return Intersection::MakeNullIntersection(*this);

    const float thc = sqrt(radius2 - d2);
    float t0 = tca - thc;
    float t1 = tca + thc;

    if (t0 > t1) {
        const float tmp = t0;
        t0 = t1;
        t1 = tmp;
    }

    if (t0 < EPSILON) {
        t0 = t1;

        if (t0 < 0)
            return Intersection::MakeNullIntersection(*this);
    }

    Intersection intersection;
    intersection.inRay    = *this;
    intersection.t        = t0;
    intersection.location = this->origin + t0 * this->direction;
    intersection.normal   = Vec4f32::Normalized3D(intersection.location - sphere.center);
    intersection.material = sphere.material;

    return intersection;
}

template <>
__device__ Intersection Ray::Intersects(const Plane& plane) const noexcept {
    const float denom = Vec4f32::DotProduct3D(plane.normal, this->direction);
    if (abs(denom) >= EPSILON) {
        const Vec4f32 v = plane.position - this->origin;
        const float t = Vec4f32::DotProduct3D(v, plane.normal) / denom;

        if (t >= 0) {
            Intersection intersection;
            intersection.inRay    = *this;
            intersection.t        = t;
            intersection.location = this->origin + t * this->direction;
            intersection.normal   = plane.normal;
            intersection.material = plane.material;

            return intersection;
        }
    }

    return Intersection::MakeNullIntersection(*this);
}

// https://en.wikipedia.org/wiki/M%C3%B6ller%E2%80%93Trumbore_intersection_algorithm#:~:text=The%20M%C3%B6ller%E2%80%93Trumbore%20ray%2Dtriangle,the%20plane%20containing%20the%20triangle.
template <>
__device__ Intersection Ray::Intersects(const Triangle& triangle) const noexcept {
    // compute plane's normal
    Vec4f32 v0v1 = triangle.p1 - triangle.p0; 
    Vec4f32 v0v2 = triangle.p2 - triangle.p0; 
    // no need to normalize
    Vec4f32 N = Vec4f32::CrossProduct3D(v0v1, v0v2); // N 
    float area2 = N.GetLength3D(); 
 
    // Step 1: finding P
 
    // check if ray and plane are parallel ?
    float NdotRayDirection = Vec4f32::DotProduct3D(N, this->direction); 
    if (fabs(NdotRayDirection) < EPSILON) // almost 0 
        return Intersection::MakeNullIntersection(*this); // they are parallel so they don't intersect ! 
 
    // compute d parameter using equation 2
    float d = Vec4f32::DotProduct3D(N, triangle.p0); 
 
    // compute t (equation 3)
    float t = (Vec4f32::DotProduct3D(N, this->origin) + d) / NdotRayDirection; 
    // check if the triangle is in behind the ray
    if (t < 0) return Intersection::MakeNullIntersection(*this); // the triangle is behind 
 
    // compute the intersection point using equation 1
    Vec4f32 P = this->origin + t * this->direction; 
 
    // Step 2: inside-outside test
    Vec4f32 C; // vector perpendicular to triangle's plane 
 
    // edge 0
    Vec4f32 edge0 = triangle.p1 - triangle.p0; 
    Vec4f32 vp0 = P - triangle.p0; 
    C = Vec4f32::DotProduct3D(edge0, vp0);
    if (Vec4f32::DotProduct3D(N, C) < 0) return Intersection::MakeNullIntersection(*this); // P is on the right side 
 
    // edge 1
    Vec4f32 edge1 = triangle.p2 - triangle.p1; 
    Vec4f32 vp1 = P - triangle.p1; 
    C = Vec4f32::DotProduct3D(edge1, vp1); 
    if (Vec4f32::DotProduct3D(N, C) < 0) return Intersection::MakeNullIntersection(*this);; // P is on the right side 
 
    // edge 2
    Vec4f32 edge2 = triangle.p0 - triangle.p2; 
    Vec4f32 vp2 = P - triangle.p2; 
    C = Vec4f32::DotProduct3D(edge2, vp2); 
    if (Vec4f32::DotProduct3D(N, C) < 0) return Intersection::MakeNullIntersection(*this);; // P is on the right side; 
 
    Intersection intersection;
    intersection.inRay    = *this;
    intersection.t        = t;
    intersection.location = this->origin + t * this->direction;
    intersection.normal   = Vec4f32{};//todo
    intersection.material = triangle.material;

    return intersection;
}

__device__ inline Ray GenerateCameraRay(const size_t& pixelX, const size_t& pixelY, const GPU_Ptr<RenderParams>& pRanderParams, hiprandState_t* const randState) noexcept {
    const RenderParams& renderParams = *pRanderParams;

    Ray ray;
    ray.origin = renderParams.camera.position;
    ray.direction = Vec4f32::Normalized3D(Vec4f32(
        (2.0f  * ((pixelX + RandomFloat(randState)) / static_cast<float>(renderParams.width))  - 1.0f) * tan(renderParams.camera.fov) * static_cast<float>(renderParams.width) / static_cast<float>(renderParams.height),
        (-2.0f * ((pixelY + RandomFloat(randState)) / static_cast<float>(renderParams.height)) + 1.0f) * tan(renderParams.camera.fov),
        1.0f,
        0.f));

    return ray;
}

template <typename T_OBJ>
__device__ inline void FindClosestIntersection(const Ray& ray,
                                               Intersection& closest, // in/out
                                               const GPU_ArrayView<T_OBJ>& objArrayView) noexcept {
    for (size_t i = 0; i < objArrayView.GetCount(); i++) {
        const Intersection current = ray.Intersects(objArrayView[i]);

        if (current.t < closest.t)
            closest = current;
    }
}

template <template<typename _IDC_T, Device _IDC_D> typename Container, Device D>
struct Primitives {
    Container<Sphere, D>   spheres;
    Container<Plane, D>    planes;
    Container<Triangle, D> triangles;

    Primitives() = default;

    template <Device D_2>
    inline Primitives(const Primitives<Container, D_2>& o) noexcept {
        this->spheres   = o.spheres;
        this->planes    = o.planes;
        this->triangles = o.triangles;
    }

    template <template<typename, Device> typename C_2, Device D_2>
    inline Primitives(const Primitives<C_2, D_2>& o) noexcept {
        this->spheres   = o.spheres;
        this->planes    = o.planes;
        this->triangles = o.triangles;
    }
}; // Primitives

__device__ Intersection
FindClosestIntersection(const Ray& ray, const Primitives<ArrayView, Device::GPU>& primitives) noexcept {
    Intersection closest;
    closest.t = FLT_MAX;
    
    FindClosestIntersection(ray, closest, primitives.spheres);
    FindClosestIntersection(ray, closest, primitives.planes);
    FindClosestIntersection(ray, closest, primitives.triangles);

    return closest;
}

template <size_t _N>
__device__ Colorf32 RayTrace(const Ray& ray,
                             const Primitives<ArrayView, Device::GPU>& primitives,
                             hiprandState_t* const randState) {
    const auto intersection = FindClosestIntersection(ray, primitives);

    if constexpr (_N < MAX_REC) {
        if (intersection.t != FLT_MAX) {
            const Material& material = intersection.material;
            
            Ray newRay;
            newRay.origin = intersection.location + EPSILON * intersection.normal;
            
            const float rngd = RandomFloat(randState);

            if (material.reflectance > rngd) {
                // Compute Reflexion
                newRay.direction = material.roughness * Random3DUnitVector(randState) + (1 - material.roughness) * Vec4f32::Reflected3D(ray.direction, intersection.normal);
            } else if (material.transparency + material.reflectance > rngd) {
                // Compute Transparency
                const bool outside = Vec4f32::DotProduct3D(ray.direction, intersection.normal) < 0;

                newRay.direction = Vec4f32::Normalized3D(Refract(ray.direction, intersection.normal, material.index_of_refraction));
                newRay.origin    = intersection.location + (outside ? -1 : 1) * EPSILON * intersection.normal;
            } else {
                // Compute Diffuse
                newRay.direction = Random3DUnitVector(randState);
            }
            
            const Colorf32 materialComp = RayTrace<_N + 1u>(newRay, primitives, randState);
            const Colorf32 finalColor   = material.emittance + material.diffuse * materialComp;

            return finalColor;
        }
    }
    
    // Black
    return Vec4f32{0.f, 0.f, 0.F, 1.f};
}

// Can't pass arguments via const& because these variables exist on the host and not on the device
__global__ void RayTracingDispatcher(const GPU_Ptr<Coloru8> pSurface,
                                     const GPU_Ptr<RenderParams> pParams,
                                     const Primitives<ArrayView, Device::GPU> primitives) {

    hiprandState_t randState;

    // Calculate the thread's (X, Y) location
    const size_t pixelX = threadIdx.x + blockIdx.x * blockDim.x;
    const size_t pixelY = threadIdx.y + blockIdx.y * blockDim.y;

    hiprand_init(pixelX, pixelY, 0, &randState);

    // Bounds check
    if (pixelX >= pParams->width || pixelY >= pParams->height) return;

    // Determine the pixel's index into the image buffer
    const size_t index = pixelX + pixelY * pParams->width;

    const Ray cameraRay = GenerateCameraRay(pixelX, pixelY, pParams, &randState);

    // the current pixel's color (represented with floating point components)
    Colorf32 pixelColor{};
    for (size_t i = 0; i < SPP; i++)
        pixelColor += RayTrace<0>(cameraRay, primitives, &randState);
    
    pixelColor *= 255.f / static_cast<float>(SPP);
    pixelColor.Clamp(0.f, 255.f);

    // Save the result to the buffer
    *(pSurface + index) = Coloru8(pixelColor.x, pixelColor.y, pixelColor.z, pixelColor.w);
}

class PolarTracer {
private:
    struct {
        RenderParams m_renderParams;
    } host;

    struct {
        Image<Coloru8, Device::GPU> m_frameBuffer;
        GPU_UniquePtr<RenderParams> m_pRenderParams;

        Primitives<Array, Device::GPU> m_primitives;
    } device;

public:
    PolarTracer(const RenderParams& renderParams, const Primitives<Array, Device::CPU>& primitives)
        : host{ renderParams }
    {
        this->device.m_frameBuffer   = Image<Coloru8, Device::GPU>(renderParams.width, renderParams.height);
        this->device.m_pRenderParams = AllocateSingle<RenderParams, Device::GPU>();
        this->device.m_primitives    = primitives;

        const auto src = CPU_Ptr<RenderParams>(&this->host.m_renderParams);
        CopySingle(this->device.m_pRenderParams, src);
    }

    inline void RayTraceScene(const Image<Coloru8, Device::CPU>& outSurface) {
        assert(outSurface.GetWidth() == this->host.m_renderParams.width && outSurface.GetHeight() == this->host.m_renderParams.height);

        const size_t bufferSize = outSurface.GetPixelCount() * sizeof(Coloru8);

        // Allocate 1 thread per pixel of coordinates (X,Y). Use as many blocks in the grid as needed
        // The RayTrace function will use the thread's index (both in the grid and in a block) to determine the pixel it will trace rays through
        const dim3 dimBlock = dim3(16, 16); // Was 32x32: 32 warps of 32 threads per block (=1024 threads in total which is the hardware limit)
        const dim3 dimGrid = dim3(std::ceil(this->host.m_renderParams.width / static_cast<float>(dimBlock.x)),
            std::ceil(this->host.m_renderParams.height / static_cast<float>(dimBlock.y)));

        // trace rays through each pixel
        RayTracingDispatcher<<<dimGrid, dimBlock>>>(this->device.m_frameBuffer.GetPtr(),
            this->device.m_pRenderParams,
            this->device.m_primitives);

        // wait for the job to finish
        printf("%s\n", hipGetErrorString(hipDeviceSynchronize()));

        // copy the gpu buffer to a new cpu buffer
        CopySize(outSurface.GetPtr(), this->device.m_frameBuffer.GetPtr(), bufferSize);
    }
}; // PolarTracer

#define WIDTH  (1920)
#define HEIGHT (1080)

int main(int argc, char** argv) {
    Image<Coloru8, Device::CPU> image(WIDTH, HEIGHT);

    RenderParams renderParams;

    renderParams.width  = WIDTH;
    renderParams.height = HEIGHT;
    renderParams.camera.position = Vec4f32(0.f, .5f, -2.f, 0.f);
    renderParams.camera.fov      = 3.141592f / 4.f;

    Primitives<Array, Device::CPU> primitives;
    primitives.spheres = CPU_Array<Sphere>(2);
    primitives.planes  = CPU_Array<Plane>(5);
    primitives.triangles = CPU_Array<Triangle>(1);

    for (size_t i = 0; i < primitives.spheres.GetCount(); ++i) {
        auto& o = primitives.spheres[i];
        o.material.reflectance = 0.f;
        o.material.roughness = 1.0f;
        o.material.transparency = 0.f;
        o.material.index_of_refraction = 1.0f;
    }

    for (size_t i = 0; i < primitives.planes.GetCount(); ++i) {
        auto& o = primitives.planes[i];
        o.material.reflectance = 0.f;
        o.material.roughness = 1.0f;
        o.material.transparency = 0.f;
        o.material.index_of_refraction = 1.0f;
    }

    for (size_t i = 0; i < primitives.triangles.GetCount(); ++i) {
        auto& o = primitives.triangles[i];
        o.material.reflectance = 0.f;
        o.material.roughness = 1.0f;
        o.material.transparency = 0.f;
        o.material.index_of_refraction = 1.0f;
    }

    primitives.spheres[0].center = Vec4f32{ 0.0f, 1.5f, 0.5f, 0.f };
    primitives.spheres[0].radius = 0.5f;
    primitives.spheres[0].material.diffuse   = Colorf32{ 1.f, 1.f, 1.f, 1.f };
    const float li = 5.f;
    primitives.spheres[0].material.emittance = Colorf32{ li, li, li, 1.f };

    primitives.spheres[1].center = Vec4f32{ 0.0f, 0.3f, 1.0f, 0.f };
    primitives.spheres[1].radius = 0.5f;
    primitives.spheres[1].material.diffuse   = Colorf32{ 1.f, 0.6f, 0.3f, 1.f };
    primitives.spheres[1].material.emittance = Colorf32{ 0.f, 0.f, 0.f, 1.f };
    primitives.spheres[1].material.reflectance = 0.1f;

    primitives.planes[0].position = Vec4f32{ 0.f, -.25f, 0.f, 0.f};
    primitives.planes[0].normal   = Vec4f32{ 0.f, 1.f, 0.f, 0.f};
    primitives.planes[0].material.diffuse   = Colorf32{1.f, 1.f, 1.f, 1.f};
    primitives.planes[0].material.emittance = Colorf32{0.f, 0.f, 0.f, 1.f};

    primitives.planes[1].position = Vec4f32{ 0.f, 0.f, 1.f, 0.f};
    primitives.planes[1].normal   = Vec4f32{ 0.f, 0.f, -1.f, 0.f};
    primitives.planes[1].material.diffuse   = Colorf32{0.75f, 0.75f, 0.75f, 1.f};
    primitives.planes[1].material.emittance = Colorf32{0.f, 0.f, 0.f, 1.f};
    primitives.planes[1].material.reflectance = 1.f;
    primitives.planes[1].material.roughness   = 0.f;

    primitives.planes[2] = primitives.planes[1];
    primitives.planes[2].position = Vec4f32{1.f, 0.f, 0.f, 0.f};
    primitives.planes[2].normal   = Vec4f32{-1.f, 0.f, 0.f, 0.f};
    primitives.planes[2].material.roughness   = 0.f;
    primitives.planes[2].material.reflectance = 0.8f;

    primitives.planes[3] = primitives.planes[1];
    primitives.planes[3].position = Vec4f32{-1.f, 0.f, 0.f, 0.f};
    primitives.planes[3].normal   = Vec4f32{1.f, 0.f, 0.f, 0.f};
    primitives.planes[3].material.roughness = 0.25f;

    primitives.planes[4].position = Vec4f32{ 0.f, 0.f, renderParams.camera.position.z - 1.f, 0.f};
    primitives.planes[4].normal   = Vec4f32{ 0.f, 0.f, 1.f, 0.f};
    primitives.planes[4].material.diffuse   = Colorf32{.75f, .75f, .75f, 1.f};
    primitives.planes[4].material.emittance = Colorf32{0.f, 0.f, 0.f, 1.f};

    primitives.triangles[0].p2 = Vec4f32{ -0.5f, 1.f, 0.5f, 0.f};
    primitives.triangles[0].p1 = Vec4f32{ 0.5f, 1.f, 0.5f, 0.f};
    primitives.triangles[0].p0 = Vec4f32{ -0.5f, 0.f, 0.5f, 0.f};
    primitives.triangles[0].material.diffuse   = Colorf32{1.f, 0.f, 0.f, 1.f};
    primitives.triangles[0].material.emittance = Colorf32{0.f, 0.f, 0.f, 1.f};

    PolarTracer pt(renderParams, primitives);

    const auto startTime = std::chrono::high_resolution_clock::now();
    pt.RayTraceScene(image);
    const auto endTime   = std::chrono::high_resolution_clock::now();

    const double duration = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count() / 1000.f;

    std::cout << std::fixed << '\n';
    std::cout << "Took " << duration << "s to render a " << WIDTH << " by " << HEIGHT << " image at " << SPP << " SPP with a maximum recursion depth of " << MAX_REC << ".\n";
    std::cout << "Big numbers:\n";
    std::cout << "-->" << (unsigned int)(WIDTH * HEIGHT) << " pixels.\n";
    std::cout << "-->" << (unsigned int)(WIDTH * HEIGHT * SPP) << " samples.\n";
    std::cout << "-->" << (unsigned int)(WIDTH * HEIGHT * SPP * MAX_REC) << " photons.\n";
    std::cout << "Timings:\n";
    std::cout << "-->" << (unsigned int)((WIDTH * HEIGHT * SPP) / duration) << " samples per sec.\n";
    std::cout << "-->" << (unsigned int)((WIDTH * HEIGHT * SPP * MAX_REC) / duration) << " photon paths per sec.\n";

    SaveImage(image, "frame");
}
